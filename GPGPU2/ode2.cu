
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>

struct LVState
{
    double rabbits, wolves;
};

__host__ __device__ int size( LVState const& ){ return 2; }

template<typename F> __device__ LVState map(F f, LVState const& s){ return LVState{ f(s.rabbits), f(s.wolves) }; }
template<typename F> __device__ LVState zip(F f, LVState const& s, LVState const& s2){ return LVState{ f(s.rabbits, s2.rabbits), f(s.wolves, s2.wolves) }; }
template<typename F> __device__ auto reducel(F f, LVState const& s){ return f(s.rabbits, s.wolves); }

struct RungeKutta4Stepper
{
    double atol, rtol;

    template<typename TTime, typename TStep, typename TState, typename TRHS> __device__ 
    TState operator()( TTime time, TStep h, TState state, TRHS rhs, double* err) const
    {
        const auto scl_mul  = []__device__ (auto scl){ return [=]__device__ (auto x){ return scl*x; }; };
        const auto add      = []__device__ (auto const& x, auto const& y) { return x + y; };
        const auto rel_diff = [&]__device__ (auto const& x, auto const& y)
        {
           auto scale = atol + rtol * max(x, y);
           return (x-y)*(x-y)/scale/scale;
        };

        TState k1 = rhs(time, state);
        TState k2 = rhs(time + 0.5 * h, zip(add, state, map( scl_mul(h*0.5), k1 )));
        TState k3 = rhs(time + 0.5 * h, zip(add, state, map( scl_mul(h*0.5), k2 )));
	TState k4 = rhs(time + h,       zip(add, state, map( scl_mul(h),     k3 )));
	
        TState sum_state = zip(add, zip(add, k1, k4), map( scl_mul(2.0), zip(add, k2, k3)));
	TState res1 = zip(add, state, map( scl_mul(h/6.0), sum_state ) );//RK4 step
        TState res0 = zip(add, state, map( scl_mul(h), k1 ) );//Euler step

        *err = sqrt(reducel(add, zip(rel_diff, res1, res0)) / size(state));
        return res1;
    }
};

template<typename Stepper, typename T, typename H, typename RHS, typename S>
__global__ void step_impl(Stepper stepper, T t, int max_steps, H h0, RHS rhs, S* src, S* path, T* path0)
{
    auto i = blockIdx.x*blockDim.x+threadIdx.x;
    S s[2];
    int idx = 0;
    s[idx] = src[i];
    T time = t;
    H h = h0;

    double err = 1.0, lerr = 1e-4;

    int step = 0;
    while(step < max_steps)
    {
        do{
	    s[1-idx] = stepper(time, h, s[idx], rhs, &err);
            //if(i == 128 && err > 0.9){ printf("R %e %e %e\n", time, h, err); }
            
	    h = 0.95 * h * pow(err, -(1.0/4.0 - 0.75*0.4/4.0)) * pow(lerr, 0.4/4.0);         
            lerr = err;
        }while(err > 0.9);

        //if(i == 128){ printf("A %e %e %e\n", time, h, err); }	
        __syncthreads();
	path[step * blockDim.x * gridDim.x + i] = s[1-idx];
	path0[step * blockDim.x * gridDim.x + i] = time;
	step += 1;
        idx = 1 - idx;
        time = time + h;
    }
}

template<typename Stepper, typename T, typename H, typename S, typename RHS>
std::pair<std::vector<T>, std::vector<S>> step(Stepper stepper, T t, int max_steps, H h, std::vector<S>const& src, RHS rhs)
{
    size_t n = src.size();
    static const size_t blockSize = 256;
           const size_t gridSize  = (size_t)ceil((float)n/blockSize);
    std::vector<S> resS(n*max_steps);
    std::vector<T> resT(n*max_steps);

    S* d_src;
    S* d_resS;
    T* d_resT;

    // Allocate memory for each vector on GPU
    hipMalloc(&d_src,  n*sizeof(S));
    hipMalloc(&d_resS, n*sizeof(S)*max_steps);
    hipMalloc(&d_resT, n*sizeof(T)*max_steps);

    // Copy host vectors to device
    hipMemcpy( d_src, src.data(), n*sizeof(S), hipMemcpyHostToDevice);

    //Measure time:
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    step_impl<<<gridSize, blockSize>>>(stepper, t, max_steps, h, rhs, d_src, d_resS, d_resT);
    hipEventRecord(stop);    
    
    hipEventSynchronize(stop);    
    float cuda_time = 0.0f;//msec
    hipEventElapsedTime(&cuda_time, start, stop);
    std::cout << "Elapsed time is: " << cuda_time << " msec\n";
    
    hipMemcpy( resS.data(), d_resS, n*sizeof(S)*max_steps, hipMemcpyDeviceToHost );
    hipMemcpy( resT.data(), d_resT, n*sizeof(T)*max_steps, hipMemcpyDeviceToHost );

    hipFree(d_src);
    hipFree(d_resS);
    hipFree(d_resT);
    
    return std::make_pair(resT, resS);	
}

int main()
{
	using State = LVState;

	//Lotka-Volterra
	double a = 0.5, b = 0.1, c = 0.8, d = 0.3;

	RungeKutta4Stepper rk4{1e-3, 1e-3};

	// Size of vectors
	size_t n = 1024;

	// State vectors
	std::vector<State> initial_state(n);

	// Initialize vectors on host
	for(int i = 0; i < n; i++ )
	{
            initial_state[i].rabbits = i*0.05;
            initial_state[i].wolves  = i*0.01;
	}

	auto rhs = [=]__device__ (double t, State const& s)
        {
             return LVState{ a*s.rabbits - b*s.rabbits*s.wolves, d*s.rabbits*s.wolves - c*s.wolves };
        };

	auto res = step(rk4, 0.0, 2048*4, 1e-1, initial_state, rhs);

        {
          auto N = res.first.size() / n;
	  std::ofstream file("lv.txt");
	  for(decltype(N) i=0; i<N; i++)
 	  {
            file << res.first[i*n+128] << "   " << res.second[i*n+128].rabbits << "   " << res.second[i*n+128].wolves << "\n";
          }
		//std::cout << "result[" << i << "] = " << res[i].rabbits << ", " << res[i].wolves << "\n";
	}

	return 0;
}
